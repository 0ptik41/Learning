// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// Includes CUDA
#include <hip/hip_runtime.h>
// Utilities and timing functions
#include <helper_functions.h>  

#define MAX_EPSILON_ERROR 5e-3f

// Texture reference
texture<float, 2, hipReadModeElementType> tex;

const char *imageFilename = "";


void usage(char* s){
	printf("Usage %s [image]\n", s);
}


////// Main Function //////
int main(int argc, char** argv){
	// Get Name of file to load into GPU 
	if (argc < 2){
		usage(argv[1]);
	}
	// Check Image Exists 
	char *imPath = sdkFindFilePath(argv[1], argv[0]);
	if (imPath == NULL){
		printf('[!] Cannot open %s\n', &argv[1]);
		exit(EXIT_FAILURE);
	}
	// Setup constants
	unsigned int width, height; // Image Dimensions
	float *iData = NULL; 		// Image Data 

	// Load Image into Memory
	sdkLoadPGM(argv[1], &iData, &width, &height);
	unsigned int imageSize = width * height *sizeof(float);
	printf('[*] %s Loaded\n', &argv[1]);
	printf('\t-Image Size: %d\n', imageSize);
	printf('\t-Dimensions: [%d x %d]\n', width, height);
	return 0;
}